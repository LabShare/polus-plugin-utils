#include "hip/hip_runtime.h"
/**
 * This code is an implementation of the algorithm presented by Dong et al., 2012,
 *"Efficient K-Nearest Neighbor Graph Construction for Generic Similarity Measures" 
 * and the performance has been improved by CUDA (GPU) directives.
 * @author:   Mahdi Maghrebi <mahdi.maghrebi@nih.gov>
 * March 2020
 */

#include <vector>
#include <iostream> 
#include <list>
#include <string>
#include <math.h>
#include <fstream>
#include <float.h>
#include <boost/filesystem.hpp> 
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

using namespace std;

/**
 * The Max number of Threads per Block. This is one of GPU hardware characteristics.
 */
#define MAXTPB 1024

/**
 * The Minimum number of computations that is needed to switch to GPU device (Otherwise stay in host)
 */
#define MinimumThreads 10

/**
 * Error handling for GPU Code
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/**
 * GPU Kernel definition
 */
__global__ void ComputeDistancesKernel(int * device_New_Final_List_1D, int * device_New_Final_List_Index, int Dim, double * device_New_Final_List_Dist_1D, double * device_dataPointsGPU, int * device_New_Final_List_Dist_Index){

	int localDim=Dim;
	double localvalue=0;
	int Cnts=device_New_Final_List_Index[blockIdx.x+1]-device_New_Final_List_Index[blockIdx.x];
	int Cnts_Dist=device_New_Final_List_Dist_Index[blockIdx.x+1]-device_New_Final_List_Dist_Index[blockIdx.x];	
	int par1, par2;
	int cnt=0;
	int flag=0;

	if (threadIdx.x < Cnts_Dist){
		for (int i=0; i < Cnts; ++i){
			if (flag ==1) break;
			for (int j=i+1; j < Cnts; ++j){
				if (threadIdx.x == cnt) {
					par1 = device_New_Final_List_1D[i + device_New_Final_List_Index[blockIdx.x]];
					par2 = device_New_Final_List_1D[j + device_New_Final_List_Index[blockIdx.x]]; 
					flag=1;
					break;         
				}
				++cnt;
			}
		}

		for (int i=0; i<localDim; ++i){
		    double tmp=device_dataPointsGPU[par1*localDim+i] - device_dataPointsGPU[par2*localDim+i];
		    localvalue += tmp*tmp;
			//localvalue += pow((device_dataPointsGPU[par1*localDim+i] - device_dataPointsGPU[par2*localDim+i]), 2);
		}	

		int IndexIDWrite= device_New_Final_List_Dist_Index[blockIdx.x]+threadIdx.x;
		device_New_Final_List_Dist_1D[IndexIDWrite] = localvalue;	
	}
	return;
}												

/**
 * Read the output of linux command execution 
 * @param  cmd  is the linux command to be executed
 * @return the output from the execution of the linux command
 */
std::string exec(const char* cmd) {
	std::array<char, 128> buffer;
	std::string result;
	std::unique_ptr<FILE, decltype(&pclose)> pipe(popen(cmd, "r"), pclose);
	if (!pipe) {
		throw std::runtime_error("popen() failed!");
	}
	while (fgets(buffer.data(), buffer.size(), pipe.get()) != nullptr) {
		result += buffer.data();
	}
	return result;
}

/**
 * Replace the farthest point in B_Index (for u1) with u2 if u2 is closer
 * <p>
 * This method corresponds to UPDATENN(B[u1],<u2,l,true>) in the paper
 * </p>
 * @param  Dist  represents B_Dist
 * @param  Index represents B_Index
 * @param  IsNew represents B_IsNew
 * @param  u1    the indice of point that we want to potentially update its K-NN with the point u2
 * @param  u2    the indice of potential K-NN fpr point u1
 * @param  distance the spatial distance between u1 and u2
 * @param  flag updates B_IsNew
 * @return 1 if B_Index[u1][.] is updated, 0 otherwise
 */
int UpdateNN (int** B_Index, double ** B_Dist, short** B_IsNew, short* allEntriesFilled, int K, int u1, int u2, double distance, int flag = 1) {

	if(allEntriesFilled[u1]==0){		
		for (int j = 0; j < K; j++) {	
			if (B_Dist[u1][j] < 0) {

				for (int jj = 0; jj < j; jj++) {if (B_Index[u1][jj] == u2) return 0;}

				B_Dist[u1][j] = distance;
				B_Index[u1][j] = u2;
				B_IsNew[u1][j] = flag;
				if (j==K-1) allEntriesFilled[u1]=1;
				return 1;}
		}
	}

	else{
		for (int j = 0; j < K; j++) {
			if (B_Index[u1][j] == u2) return 0;
		}

		double max = DBL_MIN;
		int index = -1;
		for (int j = 0; j < K; j++) {
			if (B_Dist[u1][j] > max) {
				max = B_Dist[u1][j];
				index = j;
			}
		}
		if (index == -1) { cout << "Error"<<endl; } 
		if (distance < max) {
			B_Dist[u1][index] = distance;
			B_Index[u1][index] = u2;
			B_IsNew[u1][index] = flag;
			return 1;
		}
		else { return 0; }
	}
	return 0;  
}


int main(int argc, char * const argv[]) {
	/**
	 * The errors and informational messages are outputted to the log file 
	 */
	ofstream logFile;
	string logFileName="Setting.txt";
	logFile.open(logFileName);

	/**
	 * The input parameters are read from command line which are as follow.
	 * inputPath: The full path to the input file containig the dataset.	 
	 * outputPath: The full path to the output csv files.
	 * K: K in K-NN that means the desired number of Nearest Neighbours to be computed.
	 * sampleRate: The rate at which we do sampling. This parameter plays a key role in the performance.
	 * This parameter is a trades-off between the performance and the accuracy of the results.
	 * Values closer to 1 provides more accurate results but the execution takes longer.	 
	 * convThreshold: Convergance Threshold. A fixed integer is used here instead of delta*N*K.	
	 * colIndex1 and colIndex2 (optional): The indices of columns from the input csv file where raw data exists continuously in between.
	 * If these two arguments were left blank, the code assumes that the entire input csv file is raw data 
	 * and automatically computes the number of columns in the input csv file. 	
	 */	 
	string filePath, outputPath, outputPath2, inputPath,LogoutputPath;
	int K,convThreshold, colIndex1=-1, colIndex2=-1;
	float sampleRate;	

	for (int i=1; i<argc;++i){
		if (string(argv[i])=="--inputPath") {
			inputPath=argv[i+1];

			if(!boost::filesystem::exists(inputPath) || !boost::filesystem::is_directory(inputPath))
			{
				logFile << "Incorrect input path";
				cout << "Incorrect input path";
				return 1;
			}

			const std::string ext = ".csv";
			boost::filesystem::recursive_directory_iterator it(inputPath);
			boost::filesystem::recursive_directory_iterator endit;

			bool fileFound = false;
			while(it != endit) {
				if(boost::filesystem::is_regular_file(*it) && it->path().extension() == ext){
					fileFound = true;
					filePath = it->path().string();
					break;
				}
				++it;
			}
			if (!fileFound){
				logFile << "CSV file is not found in the input path";
				cout << "CSV file is not found in the input path";
				return 1;
			}
		}
		else if (string(argv[i])=="--K") K=atoi(argv[i+1]);
		else if (string(argv[i])=="--sampleRate") sampleRate=stof(argv[i+1]);
		else if (string(argv[i])=="--convThreshold") convThreshold=stof(argv[i+1]);
		else if (string(argv[i])=="--outputPath"){
			boost::filesystem::path p(argv[i+1]);

			if(!boost::filesystem::exists(p) || !boost::filesystem::is_directory(p))
			{
				logFile << "Incorrect output path";
				cout << "Incorrect output path";
				return 1;
			}

			LogoutputPath=argv[i+1];
			boost::filesystem::path joinedPath = p / boost::filesystem::path("KNN_Indices.csv");
			outputPath = joinedPath.string();
			boost::filesystem::path joinedPath2 = p / boost::filesystem::path("KNN_Distances.csv");
			outputPath2 = joinedPath2.string();
		}
		else if (string(argv[i])=="--colIndex1") {
			colIndex1=stof(argv[i+1]);
			if (colIndex1<1) {
				logFile << "colIndex1 should be greater than 1";
				cout << "colIndex1 should be greater than 1";
				return 1;
			}
		}
		else if (string(argv[i])=="--colIndex2") {
			colIndex2=stof(argv[i+1]); 
			if (colIndex2<1) {
				logFile << "colIndex2 should be greater than 1";
				cout << "colIndex2 should be greater than 1";
				return 1;
			}
		}    
	}	

	logFile<<"------------The following Input Arguments were read------------"<<endl;
	logFile<<"The full path to the input file: "<< filePath<<endl;	
	logFile<<"The desired number of NN to be computed: "<< K <<endl;
	logFile<<"The sampleRate(The rate at which we do sampling): "<< sampleRate <<endl; 
	logFile<<"The convergance threshold: "<< convThreshold <<endl; 				
	logFile<<"The full path to the output file1: "<< outputPath<<endl;
	logFile<<"The full path to the output file2: "<< outputPath2<<endl;
	if (colIndex1 != -1) {logFile<<"The optioanl column index starts from: "<< colIndex1<<endl;}	
	if (colIndex2 != -1) {logFile<<"The optioanl column index ends at: "<< colIndex2<<endl;}		

	cout<<"------------The following Input Arguments were read------------"<<endl;	
	cout<<"The full path to the input file: "<< filePath<<endl;
	cout<<"The desired number of NN to be computed: "<< K <<endl;	
	cout<<"The sampleRate(The rate at which we do sampling): "<< sampleRate <<endl; 	
	cout<<"The convergance threshold: "<< convThreshold <<endl; 	
	cout<<"The full path to the output file1: "<< outputPath<<endl;
	cout<<"The full path to the output file2: "<< outputPath2<<endl;
	if (colIndex1 != -1) {cout<<"The optioanl column index starts from: "<< colIndex1<<endl;}	
	if (colIndex2 != -1) {cout<<"The optioanl column index ends at: "<< colIndex2<<endl;}	

	/**
	 * Size of Dataset without the header (i.e.(#Rows in dataset)-1).
	 */	
	string cmd="wc -l "+filePath;
	string outputCmd = exec(cmd.c_str());
	const int N=stoi(outputCmd.substr(0, outputCmd.find(" ")))-1;
	/**
	 * Dimension of Dataset (#Columns)
	 * is computed automatically if not passed as argument in command line
	 * otherwise, the range (beginning and end) for the column index of input csv file is needed
	 */
	int Dim;
	string cmd2="head -n 1 "+ filePath + " |tr '\\,' '\\n' |wc -l ";
	Dim = stoi(exec(cmd2.c_str())); 

	logFile<<"The input csv file contains "<<N<<" rows of raw data with "<< Dim<< " columns(features)"<<endl; 
	cout<<"The input csv file contains "<<N<<" rows of raw data with "<< Dim<< " columns(features)"<<endl; 

	/**
	 * A 2D Array containing the entire input dataset (read from filePath).
	 */
	double** dataPoints = new double*[N];
	for (int i = 0; i < N; ++i) { dataPoints[i] = new double[Dim]; }

	double* dataPointsGPU = new double[N*Dim];
	/**
	 * indices of K-NN for all the points in dataset
	 */
	int** B_Index = new int*[N];
	for (int i = 0; i < N; ++i) { B_Index[i] = new int[K]; }
	/**
	 * corresponding distance for K-NN indices stored in B_Index
	 */
	double** B_Dist = new double*[N];
	for (int i = 0; i < N; ++i) { B_Dist[i] = new double[K]; }
	/**
	 * corresponding flag for K-NN indices stored in B_Index
	 */
	short** B_IsNew = new short*[N];
	for (int i = 0; i < N; ++i) { B_IsNew[i] = new short[K]; }
	/**
	 * Data structure for new[v]
	 */
	vector<int> *New_Index = new std::vector<int>[N];
	/**
	 * Data structure for REVERSE(new[v]) or new'
	 */
	vector<int> *Reverse_New_Index = new vector<int>[N];
	/**
	 * Data Structure for SAMPLE(new'[v],pk)
	 */
	vector<int> *Sampled_Reverse_New_Index = new vector<int>[N];
	/**
	 * Data Structure for new[v] U SAMPLE(new'[v],pk)
	 */
	vector<int> *New_Final_List = new vector<int>[N];
	/**
	 * An approximation of zero in computing distances. Two points with the distance
	 * smaller than epsilon are considered as one point.
	 */
	double epsilon = 1e-10; //
	short* allEntriesFilled = new short[N];
	/**
	 * At first, let's Read Dataset from Input File
	 */
	ifstream infile;
	infile.open(filePath);
	if (infile.fail())
	{
		logFile << "error in Opening Input File" << endl;
		cout << "error in Opening Input File" << endl;
		return 1;
	}
	/**
	 * Remove the header info
	 */
	string dummyLine;
	getline(infile, dummyLine);
	/**
	 * Reading the Entire Dataset
	 */
	if (argc==11){
		for (int i = 0; i < N; ++i) {
			string temp, temp2;
			getline(infile, temp);
			for (int j = 0; j < Dim; ++j) {
				temp2 = temp.substr(0, temp.find(","));
				double tempV=atof(temp2.c_str());
				dataPoints[i][j] = tempV;
				dataPointsGPU[i*Dim+j] = tempV;
				temp.erase(0, temp.find(",") + 1);
			}
		}
	} else {
		for (int i = 0; i < N; ++i) {
			string temp, temp2;
			getline(infile, temp);
			for (int j = 0; j < Dim; ++j) {
				temp2 = temp.substr(0, temp.find(","));
				if (j >= colIndex1-1 && j < colIndex2) {
				double tempV=atof(temp2.c_str());
				dataPoints[i][j] = tempV;
				dataPointsGPU[i*Dim+j] = tempV;
				}
				temp.erase(0, temp.find(",") + 1);
			}
		}	
	}
	infile.close();

	if (colIndex1 != -1) Dim=colIndex2-colIndex1+1;
	if (Dim <1) {
		logFile << "Error in Computing the Dimension of input csv file" << endl;
		cout << "Error in Computing the Dimension of input csv file" << endl;
		return 1;	
	}
	//Convert Pagged Memory to the Pinned Memory for better performance
	hipHostRegister(dataPointsGPU,N*Dim*sizeof(double),0);
	/**
	 * Copy the GPU version of input data (dataPointsGPU) to GPU memory (device_dataPointsGPU)
	 */
	hipStream_t stream;
	hipStreamCreate(&stream);

	double * device_dataPointsGPU;
	hipMalloc ((void **) &device_dataPointsGPU, N*Dim*sizeof(double));            
	hipMemcpyAsync (device_dataPointsGPU, dataPointsGPU, N*Dim*sizeof(double),hipMemcpyHostToDevice, stream); 	
	gpuErrchk(hipPeekAtLastError());	

	/**
	 * define a seed for random generator. Using a constant value produces
	 * the same set of random numbers and is good for debugging. Alternatively,
	 * we can select the seed number randomly as srand(time(NULL))
	 */
	srand(17);
	/**
	 * Initialization of Arrays B_IsNew and B_Dist
	 */
	for (int i = 0; i < N; ++i) {
		allEntriesFilled[i]=0;
		for (int j = 0; j < K; ++j) {
			B_IsNew[i][j] = 1;
			B_Dist[i][j] = -1.0;
		}
	}
	/**
	 * Random Initialization of B_Index
	 */
	int randomIndex, iter;
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < K; ++j) {
			iter = 1;
			while (iter) {
				randomIndex = rand() % N;
				if (randomIndex != i) {
					B_Index[i][j] = randomIndex;
					iter = 0;
				}
			}
		}
	}

	/**
	 * Main Loop of the Algorithm
	 */
	bool iterate = true;
	while (iterate) {
		int c_criteria = 0;
		int abort=0;
		/**
		 * Create "New" for each Datapoint
		 */
		for (int i = 0; i < N; ++i) {
			for (int j = 0; j < K; ++j) {
				if (float(rand() % 100) < sampleRate*100) {
					if (B_IsNew[i][j] == 1) {
						New_Index[i].push_back(B_Index[i][j]);
						B_IsNew[i][j] = 0;
					}
				}
			}
		}
		/**
		 * Create "New'"(or REVERSE("New")) for each Datapoint
		 */
		for (int i = 0; i < N; ++i) {
			for (int j = 0; j < New_Index[i].size(); ++j) {
				Reverse_New_Index[New_Index[i][j]].push_back(i);
			}
		}
		/**
		 * Random Sampling from "New'"
		 */
		for (int i = 0; i < N; ++i) {
			for (int j = 0; j < Reverse_New_Index[i].size(); ++j) {
				if (float(rand() % 100) < sampleRate*100) {
					Sampled_Reverse_New_Index[i].push_back(Reverse_New_Index[i][j]);
				}
			}
		}
		/**
		 * "New"= "New" U SAMPLE("New'", pK)
		 */
		for (int i = 0; i < N; ++i) {
			for (int j = 0; j < New_Index[i].size(); ++j) {
				New_Final_List[i].push_back(New_Index[i][j]);
			}
			for (int j = 0; j < Sampled_Reverse_New_Index[i].size(); ++j) {
				New_Final_List[i].push_back(Sampled_Reverse_New_Index[i][j]);
			}
		}
		/**
		 * Remove duplicates from New_Final_List
		 */
		for (int i = 0; i < N; ++i) {	
			sort(New_Final_List[i].begin(), New_Final_List[i].end());
			auto last = std::unique(New_Final_List[i].begin(), New_Final_List[i].end());
			New_Final_List[i].erase(last, New_Final_List[i].end());
		}

		/**
		 * Max_New_Final_List_Length is the maximum length of New_Final_List array
		 */
		int Max_New_Final_List_Length=0;

		for (int i = 0; i < N; ++i) {       
			if (New_Final_List[i].size()> Max_New_Final_List_Length) Max_New_Final_List_Length=New_Final_List[i].size();
		}
		/**
		 * ThreadsPerBlockNeeded is the required number of threads per block to compute the longest array of New_Final_List
		 */
		int ThreadsPerBlockNeeded=0;	
		for (int i = 0; i < Max_New_Final_List_Length; ++i) {              
			for (int j = i+1; j < Max_New_Final_List_Length; ++j) {				        
				++ThreadsPerBlockNeeded;	
			}
		}

		/**
		 * Switch to GPU computations if the following conditions met. Otherwise proceed to CPU computations. 
		 */		 
		if (ThreadsPerBlockNeeded < MAXTPB  && ThreadsPerBlockNeeded > MinimumThreads) { 
			/**
			 * TotalCounts is the total number of elements in New_Final_List
			 */		
			int TotalCounts=0;		
			for (int i = 0; i < N; ++i) {       
				TotalCounts += New_Final_List[i].size();
			}	
			/**
			 * New_Final_List_1D is the 1D representation of New_Final_List for transferring to GPU
			 */										
			int * New_Final_List_1D = new int [TotalCounts]; 
			int cnt=0;

			for (int i = 0; i < N; ++i) {
				for (int j = 0; j < New_Final_List[i].size(); ++j) {	
					New_Final_List_1D[cnt] = New_Final_List[i][j];
					++cnt;
				}
			}	
			/**
			 * device_New_Final_List_1D is on the GPU memory and contains New_Final_List_1D
			 */	
			int *device_New_Final_List_1D;	
			hipMalloc ((void **) &device_New_Final_List_1D, TotalCounts*sizeof(int)); 
			gpuErrchk(hipMemcpy (device_New_Final_List_1D, New_Final_List_1D, TotalCounts* sizeof(int),hipMemcpyHostToDevice)); 
			/**
			 * New_Final_List_Index is the index of New_Final_List[i] data. It is needed as New_Final_List has variable size in each row of data.
			 */									 
			int * New_Final_List_Index = new int [N+1];
			New_Final_List_Index[0] = 0;
			for (int i = 1; i < N+1; ++i) {	
				New_Final_List_Index[i] = New_Final_List[i-1].size()+New_Final_List_Index[i-1];
			}
			/**
			 * device_New_Final_List_Index is on the GPU memory and contains New_Final_List_Index
			 */	
			int *device_New_Final_List_Index;		
			hipMalloc ((void **) &device_New_Final_List_Index, (N+1)*sizeof(int)); 
			gpuErrchk(hipMemcpy (device_New_Final_List_Index, New_Final_List_Index, (N+1)* sizeof(int),hipMemcpyHostToDevice));
			/**
			 * New_Final_List_Dist_Index is the index of pairs of distances computed in GPU. 
			 */							     	       	
			int * New_Final_List_Dist_Index = new int [N+1];         
			int TotalCounts_Dist=0;

			for (int i = 0; i < N; ++i) {
				New_Final_List_Dist_Index[i]=TotalCounts_Dist;
				for (int j = 0; j < New_Final_List[i].size(); ++j) {	
					for (int k = j+1; k < New_Final_List[i].size(); ++k) {	
						++TotalCounts_Dist;
					}
				}				
			}
			New_Final_List_Dist_Index[N]=TotalCounts_Dist;
			/**
			 * device_New_Final_List_Dist_Index is on the GPU memory and contains New_Final_List_Dist_Index
			 */	
			int * device_New_Final_List_Dist_Index;	            
			hipMalloc ((void **) &device_New_Final_List_Dist_Index, (N+1)*sizeof(int)); 		                
			gpuErrchk(hipMemcpy (device_New_Final_List_Dist_Index, New_Final_List_Dist_Index, (N+1) * sizeof(int),hipMemcpyHostToDevice));
			/**
			 * device_New_Final_List_Dist_1D is on the GPU memory and contains 1D array of pairs of distances computed in GPU.
			 */						        
			double *device_New_Final_List_Dist_1D;  
			hipMalloc ((void **) &device_New_Final_List_Dist_1D, TotalCounts_Dist*sizeof(double)); 
			/**
			 * Launch the Kernel to compute the distance computations for all pairs of the points.
			 * hipDeviceSynchronize is required to ensure data transfer to GPU memory is already finished.
			 */				        
			gpuErrchk(hipDeviceSynchronize());	

			logFile<< "Number of Blocks = "<<N<< " and Number of Threads Per Block = "<<ThreadsPerBlockNeeded<<endl;
			cout<< "Number of Blocks = "<<N<< " and Number of Threads Per Block = "<<ThreadsPerBlockNeeded<<endl;

			ComputeDistancesKernel<<<N, ThreadsPerBlockNeeded>>>(device_New_Final_List_1D,device_New_Final_List_Index, Dim,device_New_Final_List_Dist_1D, device_dataPointsGPU,device_New_Final_List_Dist_Index);
			gpuErrchk(hipDeviceSynchronize());	
			/**
			 * New_Final_List_Dist_1D is on the host containing device_New_Final_List_Dist_1D
			 */				
			double * New_Final_List_Dist_1D = new double [TotalCounts_Dist]; 
			gpuErrchk(hipMemcpy (New_Final_List_Dist_1D, device_New_Final_List_Dist_1D, TotalCounts_Dist* sizeof(double),hipMemcpyDeviceToHost)); 

			/**
			 * Now that we have computed all the distance pairs on GPU, we update the appropriate arrays on host 
			 * c=c+UPDATENN(B[u1],<u2,l,true>)
			 */

			for (int i = 0; i < N; ++i) {
				if (abort != 0) break;
				int tmpcnt=0;

				for (int it = 0; it < New_Final_List[i].size(); ++it) {
					int par1= New_Final_List[i][it];

					for (int it2 = it+1; it2 < New_Final_List[i].size(); ++it2) {
						int par2= New_Final_List[i][it2];

						if (abort ==0) {
							double dist= New_Final_List_Dist_1D[New_Final_List_Dist_Index[i]+tmpcnt];
							double dista = sqrt(dist);
							++tmpcnt;

							if (dista < epsilon) {
								logFile << "Found Duplicate Data for Points "<< par1 << " and " << par2 <<endl;; 
								cout << "Found Duplicate Data for Points "<< par1 << " and " << par2 <<endl; 
								abort=1; iterate = false; 
							}

							c_criteria += UpdateNN(B_Index, B_Dist, B_IsNew, allEntriesFilled, K, par1, par2, dista, 1);
							c_criteria += UpdateNN(B_Index, B_Dist, B_IsNew, allEntriesFilled, K, par2, par1, dista, 1);

						}
					}
				}
			}

		/**
			 * Free the pointers' memory allocations on host and device
			 */
			 
			hipFree(device_New_Final_List_1D); 		   
			hipFree(device_New_Final_List_Index);
			hipFree(device_New_Final_List_Dist_Index);			
			hipFree(device_New_Final_List_Dist_1D);

			delete [] New_Final_List_Dist_1D, New_Final_List_Index, New_Final_List_Dist_Index;
			delete[]  New_Final_List_1D;

		} else {
			for (int i = 0; i < N; ++i) {
				if (abort != 0) break;

				for (int it = 0; it < New_Final_List[i].size(); ++it) {
					int par1= New_Final_List[i][it];

					for (int it2 = it+1; it2 < New_Final_List[i].size(); ++it2) {
						int par2= New_Final_List[i][it2];
						if (abort ==0) {

							double dist = 0;
							for (int j = 0; j < Dim; ++j) {
								dist += pow((dataPoints[par1][j] - dataPoints[par2][j]), 2);
							}

							double dista = sqrt(dist);

							if (dista < epsilon) {
								logFile << "Found Duplicate Data for Points "<< par1 << " and " << par2 <<endl;; 
								cout << "Found Duplicate Data for Points "<< par1 << " and " << par2 <<endl; 
								abort=1;iterate = false; 
							}						
							c_criteria += UpdateNN(B_Index, B_Dist, B_IsNew, allEntriesFilled, K, par1, par2, dista, 1);
							c_criteria += UpdateNN(B_Index, B_Dist, B_IsNew, allEntriesFilled, K, par2, par1, dista, 1);
						}
					}
				}
			}
		}

		logFile << "c_criteria = " << c_criteria << " With Threshold Convergence of " << convThreshold << endl;
		cout << "c_criteria = " << c_criteria << " With Threshold Convergence of " << convThreshold << endl;
		if (c_criteria < convThreshold) { iterate = false; }
		/**
		 * Clear the contents of the used data structures
		 */
		for (int i = 0; i < N; ++i) {
			New_Index[i].clear();
			Reverse_New_Index[i].clear();
			Sampled_Reverse_New_Index[i].clear();
			New_Final_List[i].clear();
		}
	}
    hipHostUnregister(dataPointsGPU);
	hipFree(device_dataPointsGPU);

	/**
	 * Sort and output the results
	 */
	ofstream outputFileIndex,outputFileDistance;
	outputFileIndex.open(outputPath);	
	outputFileDistance.open(outputPath2);

	for (int i=0; i<N; ++i){
		vector<pair<double,int>> aggregateResults;
		for (int j=0; j<K; ++j){
			aggregateResults.push_back(make_pair(B_Dist[i][j], B_Index[i][j]));
		}		
		sort(aggregateResults.begin(), aggregateResults.end());	

		for (int j=0; j<K; ++j){
			if (j != K-1) {outputFileIndex<<aggregateResults[j].second<<",";}
			else {outputFileIndex<<aggregateResults[j].second<<endl; }
		}

		for (int j=0; j<K; ++j){
			if (j != K-1) {outputFileDistance<<aggregateResults[j].first<<",";}
			else {outputFileDistance<<aggregateResults[j].first<<endl; }
		}
	}

	outputFileIndex.close();
	outputFileDistance.close();
	logFile.close();
	/**
	 * copy Logfile to the file system which could be accessed outside the docker container
	 */ 
	string cmd3="cp "+ logFileName+"  "+LogoutputPath;
	string outputCmd3 = exec(cmd3.c_str());

	return 0;
}
